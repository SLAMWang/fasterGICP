#include "hip/hip_runtime.h"
#include <fast_gicp/cuda/ndt_compute_derivatives.cuh>

#include <thrust/transform_reduce.h>

namespace fast_gicp {
namespace cuda {

namespace {

__host__ __device__ float huber(float k, float x) {
  float abs_x = fabsf(x);
  return abs_x <= k ? 1.0 : k / abs_x;
}

__host__ __device__ float cauchy(float k, float x) {
  float k_sq = k * k;
  return k_sq / (k_sq + x * x);
}

// skew symmetric matrix
__host__ __device__ Eigen::Matrix3f skew_symmetric(const Eigen::Vector3f& x) {
  Eigen::Matrix3f skew = Eigen::Matrix3f::Zero();
  skew(0, 1) = -x[2];
  skew(0, 2) = x[1];
  skew(1, 0) = x[2];
  skew(1, 2) = -x[0];
  skew(2, 0) = -x[1];
  skew(2, 1) = x[0];

  return skew;
}

struct p2d_ndt_compute_derivatives_kernel {
  p2d_ndt_compute_derivatives_kernel(
    const GaussianVoxelMap& target_voxelmap,
    const thrust::device_vector<Eigen::Vector3f>& source_points,
    const thrust::device_ptr<const Eigen::Isometry3f>& x_eval_ptr,
    const thrust::device_ptr<const Eigen::Isometry3f>& x_ptr)
  : trans_eval_ptr(x_eval_ptr),
    trans_ptr(x_ptr),
    src_means_ptr(source_points.data()),
    voxelmap_info_ptr(target_voxelmap.voxelmap_info_ptr.data()),
    voxel_num_points_ptr(target_voxelmap.num_points.data()),
    voxel_means_ptr(target_voxelmap.voxel_means.data()),
    voxel_covs_ptr(target_voxelmap.voxel_covs.data()) {}

  // Here, we use simple point-to-distribution MLE distance equivalent to [Biber, IROS2003]
  // Because this formulation can be more sensitive to outliers compared to another formulation based on
  // Gaussian + uniform distribution [Magnusson, 2009], we use some robust kernels to filter out outliers
  __host__ __device__ thrust::tuple<float, Eigen::Matrix<float, 6, 6>, Eigen::Matrix<float, 6, 1>> operator()(const thrust::pair<int, int>& correspondence) const {
    const Eigen::Vector3f& mean_A = thrust::raw_pointer_cast(src_means_ptr)[correspondence.first];

    if (correspondence.second < 0) {
      return thrust::make_tuple(0.0f, Eigen::Matrix<float, 6, 6>::Zero().eval(), Eigen::Matrix<float, 6, 1>::Zero().eval());
    }

    int num_points = thrust::raw_pointer_cast(voxel_num_points_ptr)[correspondence.second];
    const Eigen::Vector3f& mean_B = thrust::raw_pointer_cast(voxel_means_ptr)[correspondence.second];
    const Eigen::Matrix3f& cov_B = thrust::raw_pointer_cast(voxel_covs_ptr)[correspondence.second];

    if (num_points <= 6) {
      return thrust::make_tuple(0.0f, Eigen::Matrix<float, 6, 6>::Zero().eval(), Eigen::Matrix<float, 6, 1>::Zero().eval());
    }

    const auto& trans_eval = *thrust::raw_pointer_cast(trans_eval_ptr);
    const auto& trans = *thrust::raw_pointer_cast(trans_ptr);

    Eigen::Matrix3f R_eval = trans_eval.linear();
    Eigen::Matrix3f R = trans.linear();
    Eigen::Vector3f t = trans.translation();

    const Eigen::Vector3f transed_mean_A = R * mean_A + t;

    Eigen::Matrix3f RCR_inv = cov_B.inverse();

    Eigen::Vector3f error = mean_B - transed_mean_A;

    float w = cauchy(thrust::raw_pointer_cast(voxelmap_info_ptr)->voxel_resolution, error.norm());
    float err = w * error.transpose() * RCR_inv * error;

    Eigen::Matrix<float, 3, 6> dtdx0;
    dtdx0.block<3, 3>(0, 0) = skew_symmetric(transed_mean_A);
    dtdx0.block<3, 3>(0, 3) = -Eigen::Matrix3f::Identity();

    Eigen::Matrix<float, 3, 6> J = dtdx0;

    Eigen::Matrix<float, 6, 6> H = w * J.transpose() * RCR_inv * J;
    Eigen::Matrix<float, 6, 1> b = w * J.transpose() * RCR_inv * error;

    return thrust::make_tuple(err, H, b);
  }

  thrust::device_ptr<const Eigen::Isometry3f> trans_eval_ptr;
  thrust::device_ptr<const Eigen::Isometry3f> trans_ptr;

  thrust::device_ptr<const Eigen::Vector3f> src_means_ptr;

  thrust::device_ptr<const VoxelMapInfo> voxelmap_info_ptr;
  thrust::device_ptr<const int> voxel_num_points_ptr;
  thrust::device_ptr<const Eigen::Vector3f> voxel_means_ptr;
  thrust::device_ptr<const Eigen::Matrix3f> voxel_covs_ptr;
};

struct d2d_ndt_compute_derivatives_kernel {
  d2d_ndt_compute_derivatives_kernel(
    const GaussianVoxelMap& target_voxelmap,
    const GaussianVoxelMap& source_voxelmap,
    const thrust::device_ptr<const Eigen::Isometry3f>& x_eval_ptr,
    const thrust::device_ptr<const Eigen::Isometry3f>& x_ptr)
  : trans_eval_ptr(x_eval_ptr),
    trans_ptr(x_ptr),
    src_means_ptr(source_voxelmap.voxel_means.data()),
    src_covs_ptr(source_voxelmap.voxel_covs.data()),
    voxelmap_info_ptr(target_voxelmap.voxelmap_info_ptr.data()),
    voxel_num_points_ptr(target_voxelmap.num_points.data()),
    voxel_means_ptr(target_voxelmap.voxel_means.data()),
    voxel_covs_ptr(target_voxelmap.voxel_covs.data()) {}

  // calculate derivatives
  __host__ __device__ thrust::tuple<float, Eigen::Matrix<float, 6, 6>, Eigen::Matrix<float, 6, 1>> operator()(const thrust::pair<int, int>& correspondence) const {
    const Eigen::Vector3f& mean_A = thrust::raw_pointer_cast(src_means_ptr)[correspondence.first];
    const Eigen::Matrix3f& cov_A = thrust::raw_pointer_cast(src_covs_ptr)[correspondence.first];

    if (correspondence.second < 0) {
      return thrust::make_tuple(0.0f, Eigen::Matrix<float, 6, 6>::Zero().eval(), Eigen::Matrix<float, 6, 1>::Zero().eval());
    }

    int num_points = thrust::raw_pointer_cast(voxel_num_points_ptr)[correspondence.second];
    const Eigen::Vector3f& mean_B = thrust::raw_pointer_cast(voxel_means_ptr)[correspondence.second];
    const Eigen::Matrix3f& cov_B = thrust::raw_pointer_cast(voxel_covs_ptr)[correspondence.second];

    if (num_points <= 6) {
      return thrust::make_tuple(0.0f, Eigen::Matrix<float, 6, 6>::Zero().eval(), Eigen::Matrix<float, 6, 1>::Zero().eval());
    }

    const auto& trans_eval = *thrust::raw_pointer_cast(trans_eval_ptr);
    const auto& trans = *thrust::raw_pointer_cast(trans_ptr);

    Eigen::Matrix3f R_eval = trans_eval.linear();
    Eigen::Matrix3f R = trans.linear();
    Eigen::Vector3f t = trans.translation();

    const Eigen::Vector3f transed_mean_A = R * mean_A + t;

    Eigen::Matrix3f RCR = R_eval * cov_A * R_eval.transpose();
    Eigen::Matrix3f RCR_inv = (cov_B + RCR).inverse();

    Eigen::Vector3f error = mean_B - transed_mean_A;

    float w = cauchy(thrust::raw_pointer_cast(voxelmap_info_ptr)->voxel_resolution, error.norm());
    float err = w * error.transpose() * RCR_inv * error;

    Eigen::Matrix<float, 3, 6> dtdx0;
    dtdx0.block<3, 3>(0, 0) = skew_symmetric(transed_mean_A);
    dtdx0.block<3, 3>(0, 3) = -Eigen::Matrix3f::Identity();

    Eigen::Matrix<float, 3, 6> J = dtdx0;

    Eigen::Matrix<float, 6, 6> H = w * J.transpose() * RCR_inv * J;
    Eigen::Matrix<float, 6, 1> b = w * J.transpose() * RCR_inv * error;

    return thrust::make_tuple(err, H, b);
  }

  thrust::device_ptr<const Eigen::Isometry3f> trans_eval_ptr;
  thrust::device_ptr<const Eigen::Isometry3f> trans_ptr;

  thrust::device_ptr<const Eigen::Vector3f> src_means_ptr;
  thrust::device_ptr<const Eigen::Matrix3f> src_covs_ptr;

  thrust::device_ptr<const VoxelMapInfo> voxelmap_info_ptr;
  thrust::device_ptr<const int> voxel_num_points_ptr;
  thrust::device_ptr<const Eigen::Vector3f> voxel_means_ptr;
  thrust::device_ptr<const Eigen::Matrix3f> voxel_covs_ptr;
};

struct sum_errors_kernel {
  using Tuple = thrust::tuple<float, Eigen::Matrix<float, 6, 6>, Eigen::Matrix<float, 6, 1>>;

  __host__ __device__ Tuple operator()(const Tuple& lhs, const Tuple& rhs) {
    return thrust::make_tuple(thrust::get<0>(lhs) + thrust::get<0>(rhs), thrust::get<1>(lhs) + thrust::get<1>(rhs), thrust::get<2>(lhs) + thrust::get<2>(rhs));
  }
};

}  // namespace

double p2d_ndt_compute_derivatives(
  const GaussianVoxelMap& target_voxelmap,
  const thrust::device_vector<Eigen::Vector3f>& source_points,
  const thrust::device_vector<thrust::pair<int, int>>& correspondences,
  const thrust::device_ptr<const Eigen::Isometry3f>& linearized_x_ptr,
  const thrust::device_ptr<const Eigen::Isometry3f>& x_ptr,
  Eigen::Matrix<double, 6, 6>* H,
  Eigen::Matrix<double, 6, 1>* b) {
  auto sum_errors = thrust::transform_reduce(
    correspondences.begin(),
    correspondences.end(),
    p2d_ndt_compute_derivatives_kernel(target_voxelmap, source_points, linearized_x_ptr, x_ptr),
    thrust::make_tuple(0.0f, Eigen::Matrix<float, 6, 6>::Zero().eval(), Eigen::Matrix<float, 6, 1>::Zero().eval()),
    sum_errors_kernel());

  if (H && b) {
    *H = thrust::get<1>(sum_errors).cast<double>();
    *b = thrust::get<2>(sum_errors).cast<double>();
  }

  return thrust::get<0>(sum_errors);
}

double d2d_ndt_compute_derivatives(
  const GaussianVoxelMap& target_voxelmap,
  const GaussianVoxelMap& source_voxelmap,
  const thrust::device_vector<thrust::pair<int, int>>& correspondences,
  const thrust::device_ptr<const Eigen::Isometry3f>& linearized_x_ptr,
  const thrust::device_ptr<const Eigen::Isometry3f>& x_ptr,
  Eigen::Matrix<double, 6, 6>* H,
  Eigen::Matrix<double, 6, 1>* b) {
  auto sum_errors = thrust::transform_reduce(
    correspondences.begin(),
    correspondences.end(),
    d2d_ndt_compute_derivatives_kernel(target_voxelmap, source_voxelmap, linearized_x_ptr, x_ptr),
    thrust::make_tuple(0.0f, Eigen::Matrix<float, 6, 6>::Zero().eval(), Eigen::Matrix<float, 6, 1>::Zero().eval()),
    sum_errors_kernel());

  if (H && b) {
    *H = thrust::get<1>(sum_errors).cast<double>();
    *b = thrust::get<2>(sum_errors).cast<double>();
  }

  return thrust::get<0>(sum_errors);
}

}  // namespace cuda
}  // namespace fast_gicp
